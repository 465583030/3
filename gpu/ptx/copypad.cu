
#include <hip/hip_runtime.h>
// Copies src (3D array, size S0 x S1 x S2) to larger dst (3D array, size D0 x D1 x D2).
// src data is offset by o0,o1,o2.
// The remainder of dst is NOT zero-padded.
// E.g.:
//	a    ->  a x
//	         x x
//
extern "C" __global__ void 
copypad(float* __restrict__  dst, int D0, int D1, int D2, 
        float* __restrict__  src, int S0, int S1, int S2, 
        int o0, int o1, int o2){

	int j = blockIdx.y * blockDim.y + threadIdx.y; // index in src slice
	int k = blockIdx.x * blockDim.x + threadIdx.x;

	if(j>=S1 || k>=S2 || j>=D1 || k>=D2){
 		return;	// out of  bounds
	}

	int J = j + o1;  // index in full src
	int K = k + o2; 
	// loop over N layers
	int N = min(S0, D0);
	for (int i=0; i<N; i++){
 		int I = i + o0; // index in full src
		dst[I*D1*D2 + J*D2 + K] = src[i*S1*S2 + j*S2 + k];
	}
} 


#include "hip/hip_runtime.h"
#include "reduce.h"
#include "atomicf.h"

#define load(i) src[i]

extern "C" __global__ void
reducemax(float *src, float *dst, float initVal, int n) {
	reduce(load, fmax, atomicFmax)
}


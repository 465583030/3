#include "hip/hip_runtime.h"
#include "reduce.h"
#include "atomicf.h"

#define load(i) src[i]

extern "C" __global__ void
reducemax(float* __restrict__ src, float* __restrict__  dst, float initVal, int n) {
	reduce(load, fmax, atomicFmax)
}


#include "hip/hip_runtime.h"
#include "reduce.h"

inline __device__ float sum(float a, float b){
	return a + b;
}

#define load(i) src[i]

extern "C" __global__ void
reducesum(float *src, float *dst, float initVal, int n) {
	reduce(load, sum, atomicAdd)
}


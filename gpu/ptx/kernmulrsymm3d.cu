
#include <hip/hip_runtime.h>
// 3D micromagnetic kernel multiplication:
//
// |Mx|   |Kxx Kxy Kxz|   |Mx|
// |My| = |Kxy Kyy Kyz| * |My|
// |Mz|   |Kxz Kyz Kzz|   |Mz|
// 
// ~kernel has mirror symmetry along Y and X-axis,
// apart form first row,
// and is only stored (roughly) half:
//
// K11, K22:
// xxxxx
// aaaaa
// bbbbb
// ....
// bbbbb
// aaaaa
//
// K12:
// xxxxx
// aaaaa
// bbbbb
// ...
// -aaaa
// -bbbb

// 3D array indexing

extern "C" __global__ void 
kernmulRSymm3D(float* __restrict__  fftMx,  float* __restrict__  fftMy,  float* __restrict__  fftMz,
               float* __restrict__  fftKxx, float* __restrict__  fftKyy, float* __restrict__  fftKzz, 
               float* __restrict__  fftKyz, float* __restrict__  fftKxz, float* __restrict__  fftKxy,
               int N0, int N1, int N2){

	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.x * blockDim.x + threadIdx.x;

	if(j>= N1 || k>=N2){
 		return;	
	}

	//int I = j*N2 + k;       // linear index for upper half of kernel
	//int I2 = (N1-j)*N2 + k; // linear index for re-use of lower half

    float Kxx, Kyy, Kzz, Kxy, Kxz, Kyz;

	for(int i=0; i<N0; i++){

		int I = i*N1*N2 + j*N2 + k;
	
		if (j < N1/2 + 1){
			Kxx = fftKxx[I];
			Kyy = fftKyy[I];
			Kzz = fftKzz[I];
			Kyz = fftKyz[I];
			Kxz = fftKxz[I];
			Kxy = fftKxy[I];
		}else{
			int I2 = i*N1*N2 + (N1-j)*N2 + k;
			Kxx = fftKxx[I2];
			Kyy = fftKyy[I2];
			Kzz = fftKzz[I2];
			Kyz = -fftKyz[I2];
			Kxz = fftKxz[I2];
			Kxy = -fftKxy[I2];
		}

  		int e = 2 * I;
		float reMx = fftMx[e  ];
		float imMx = fftMx[e+1];
		float reMy = fftMy[e  ];
		float imMy = fftMy[e+1];
		float reMz = fftMz[e  ];
		float imMz = fftMz[e+1];
		
		fftMx[e  ] = reMx * Kxx + reMy * Kxy + reMz * Kxz;
		fftMx[e+1] = imMx * Kxx + imMy * Kxy + imMz * Kxz;
		fftMy[e  ] = reMx * Kxy + reMy * Kyy + reMz * Kyz;
		fftMy[e+1] = imMx * Kxy + imMy * Kyy + imMz * Kyz;
		fftMz[e  ] = reMx * Kxz + reMy * Kyz + reMz * Kzz;
		fftMz[e+1] = imMx * Kxz + imMy * Kyz + imMz * Kzz;
	}
}


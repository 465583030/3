
#include <hip/hip_runtime.h>
// dst[i] = fac1 * src1[i] + fac2 * src2[i] + fac3 * src3[i]
extern "C" __global__ void 
madd3(float* dst,  float* src1, float fac1, float* src2, float fac2, int N, float* src3, float fac3){
	int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
	if(i < N){
		dst[i] = fac1 * src1[i] + fac2 * src2[i] + fac3 * src3[i];
	}
}


#include "hip/hip_runtime.h"
#include "float3.h"

extern "C" __global__ void
normalize(float* __restrict__ vx, float* __restrict__ vy, float* __restrict__ vz, 
          float* __restrict__ mask, float norm, int N) {

	int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
	if (i < N) {

		float3 V = {vx[i], vy[i], vz[i]};
		float msk = (mask == NULL? norm: norm*mask[i]);
		V = msk * normalized(V);
		vx[i] = V.x;
		vy[i] = V.y;
		vz[i] = V.z;
	}
}


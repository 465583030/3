#include "hip/hip_runtime.h"
#include "common_stencil.h"

#define loadm(d, i) {d.x = mx[i]; d.y = my[i]; d.z = mz[i]; d = normalized(d);}
    
extern "C" __global__ void
exchange(float* __restrict__ Hx, float* __restrict__ Hy, float* __restrict__ Hz,
         float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
         float wx, float wy, float wz,
         int N0, int N1, int N2){

	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;

	if (j >= N1 || k >= N2){
		return;
	}

	for(int i=0; i<N0; i++){

		float3 m, m1, m2, H;
		loadm(m, idx(i, j, k));
		float Bsat = len(m);
		if (Bsat == 0) { Bsat = 1; }
		
		loadm(m1, idx(clamp(i+1,N0), j, k));
		loadm(m2, idx(clamp(i-1,N0), j, k));
		H  = (wx/Bsat) * ((m1-m) + (m2-m));

		loadm(m1, idx(i, clamp(j+1,N1), k));
		loadm(m2, idx(i, clamp(j-1,N1), k));
		H += (wy/Bsat) * ((m1-m) + (m2-m));

		loadm(m1, idx(i, j, clamp(k+1,N2)));
		loadm(m2, idx(i, j, clamp(k-1,N2)));
		H += (wz/Bsat) * ((m1-m) + (m2-m));

		Hx[I] = H.x;
		Hy[I] = H.y;
		Hz[I] = H.z;
	}
}


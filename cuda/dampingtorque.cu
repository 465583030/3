#include "hip/hip_runtime.h"
#include "float3.h"

// Only the damping term of the Landau-Lifshitz torque, with alpha = 1.
extern "C" __global__ void
dampingtorque(float* __restrict__  tx, float* __restrict__  ty, float* __restrict__  tz,
              float* __restrict__  mx, float* __restrict__  my, float* __restrict__  mz, 
              float* __restrict__  hx, float* __restrict__  hy, float* __restrict__  hz, int N) {

	int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
	if (i < N) {

		float3 m = {mx[i], my[i], mz[i]};
		float3 H = {hx[i], hy[i], hz[i]};
		float3 torque = -cross(m, cross(m, H));

		tx[i] = torque.x;
		ty[i] = torque.y;
		tz[i] = torque.z;
	}
}


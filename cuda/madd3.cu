#include "hip/hip_runtime.h"
#include "mask.h"

// dst[i] = fac1 * src1[i] + fac2 * src2[i] + fac3 * src3[i]
extern "C" __global__ void
madd3(float* __restrict__ dst,
      float* __restrict__ src1, float fac1,
      float* __restrict__ src2, float fac2,
      float* __restrict__ src3, float fac3, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;

    if(i < N) {
        float s1 = loadmask(src1, i);
        float s2 = loadmask(src2, i);
        float s3 = loadmask(src3, i);
        dst[i] = (fac1 * s1) + (fac2 * s2 + fac3 * s3);
        // parens for better accuracy heun solver.
    }
}


#include "hip/hip_runtime.h"
#include "stencil.h"

// Dzyaloshinskii-Moriya interaction.
// m is normalized.

extern "C" __global__ void
adddmi(float* __restrict__ Hx, float* __restrict__ Hy, float* __restrict__ Hz,
       float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
       float dx, float dy, float dz, // DMI vector in Tesla / m
       float cx, float cy, float cz, // cell size in m
       int N0, int N1, int N2){

	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;

	if (j >= N1 || k >= N2){
		return;
	}

	for(int i=0; i<N0; i++){

		int I = idx(i, j, k);

		if (dx != 0){
			float dmzdy = diff(mz, 0, 1, 0, cy); // ∂mz / ∂y
			float dmydz = diff(my, 0, 0, 1, cz); // ∂my / ∂z
			Hx[I] += dx * (-dmzdy + dmydz); 
		}

		if (dy != 0){
			float dmzdx = diff(mz, 1, 0, 0, cx);
			float dmxdz = diff(mx, 0, 0, 1, cz);
			Hy[I] += dy * (dmzdx - dmxdz); 
		}

		if (dz != 0){
			float dmydx = diff(my, 1, 0, 0, cx);
			float dmxdy = diff(mx, 0, 1, 0, cy);
			Hz[I] += dz * (-dmydx + dmxdy); 
		}
		// note: left-handed coordinate system.
	}
}


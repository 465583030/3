#include "hip/hip_runtime.h"
#include "stencil.h"
#include "float3.h"

// Exchange + Dzyaloshinskii-Moriya interaction according to
// Bagdanov and Röβler, PRL 87, 3, 2001. eq.8 (out-of-plane symmetry breaking).
// Taking into account proper boundary conditions.
// m: normalized magnetization
// H: effective field in Tesla
// D: dmi strength / Msat, in Tesla*m
// A: Aex/Msat
extern "C" __global__ void
adddmi(float* __restrict__ Hx, float* __restrict__ Hy, float* __restrict__ Hz,
       float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
       float cx, float cy, float cz, float DL, float DH, float A, int N0, int N1, int N2) {

    int i = blockIdx.z * blockDim.z + threadIdx.z;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= N0 || j >= N1 || k >= N2) {
        return;
    }

    int I = idx(i, j, k);                        // central cell index
    float3 h = make_float3(Hx[I], Hy[I], Hz[I]); // add to H
    float3 m = make_float3(mx[I], my[I], mz[I]); // central m
    float DL_2A = (DL/(2.0f*A));
    float DH_2A = (DH/(2.0f*A));

    // z derivatives (along length)
    {
        int I1 = idx(i, j, hclamp(k+1, N2));  // right index, clamped
        int I2 = idx(i, j, lclamp(k-1));      // left index, clamped

        // DMI
        float mz1 = (k+1<N2)? mz[I1] : (m.z + (cz * DL_2A * m.x)); // right neighbor
        float mz2 = (k-1>=0)? mz[I2] : (m.z - (cz * DL_2A * m.x)); // left neighbor
        h.x -= DL*(mz1-mz2)/cz;
        // note: actually 2*D * delta / (2*c)

        float mx1 = (k+1<N2)? mx[I1] : (m.x - (cz * DL_2A * m.z));
        float mx2 = (k-1>=0)? mx[I2] : (m.x + (cz * DL_2A * m.z));
        h.z += DL*(mx1-mx2)/cz;

        // Exchange
        float3 m1 = make_float3(mx1, my[I1], mz1); // right neighbor
        float3 m2 = make_float3(mx2, my[I2], mz2); // left neighbor
        h +=  (2.0f*A/(cz*cz)) * ((m1 - m) + (m2 - m));
    }

    // y derivatives (along height)
    {
        int I1 = idx(i, hclamp(j+1, N1), k);
        int I2 = idx(i, lclamp(j-1), k);

        // DMI
        float my1 = (j+1<N1)? my[I1] : (m.y + (cy * DH_2A * m.x));
        float my2 = (j-1>=0)? my[I2] : (m.y - (cy * DH_2A * m.x));
        h.x -= DH*(my1-my2)/cy;

        float mx1 = (j+1<N1)? mx[I1] : (m.x - (cy * DH_2A * m.y));
        float mx2 = (j-1>=0)? mx[I2] : (m.x + (cy * DH_2A * m.y));
        h.y += DH*(mx1-mx2)/cy;

        // Exchange
        float3 m1 = make_float3(mx1, my1, mz[I1]);
        float3 m2 = make_float3(mx2, my2, mz[I2]);
        h +=  (2.0f*A/(cy*cy)) * ((m1 - m) + (m2 - m));
    }

    // write back, result is H + Hdmi + Hex
    Hx[I] = h.x;
    Hy[I] = h.y;
    Hz[I] = h.z;
}

// Note on boundary conditions.
//
// We need the derivative and laplacian of m in point A, but e.g. C lies out of the boundaries.
// We use the boundary condition in B (derivative of the magnetization) to extrapolate m to point C:
// 	m_C = m_A + (dm/dx)|_B * cellsize
//
// When point C is inside the boundary, we just use its actual value.
//
// Then we can take the central derivative in A:
// 	(dm/dx)|_A = (m_C - m_D) / (2*cellsize)
// And the laplacian:
// 	lapl(m)|_A = (m_C + m_D - 2*m_A) / (cellsize^2)
//
// All these operations should be second order as they involve only central derivatives.
//
//    ------------------------------------------------------------------ *
//   |                                                   |             C |
//   |                                                   |          **   |
//   |                                                   |        ***    |
//   |                                                   |     ***       |
//   |                                                   |   ***         |
//   |                                                   | ***           |
//   |                                                   B               |
//   |                                               *** |               |
//   |                                            ***    |               |
//   |                                         ****      |               |
//   |                                     ****          |               |
//   |                                  ****             |               |
//   |                              ** A                 |               |
//   |                         *****                     |               |
//   |                   ******                          |               |
//   |          *********                                |               |
//   |D ********                                         |               |
//   |                                                   |               |
//   +----------------+----------------+-----------------+---------------+
//  -1              -0.5               0               0.5               1
//                                 x


#include <hip/hip_runtime.h>
// 3D micromagnetic kernel multiplication:
//
// |Mx|   |Kxx Kxy Kxz|   |Mx|
// |My| = |Kxy Kyy Kyz| * |My|
// |Mz|   |Kxz Kyz Kzz|   |Mz|
//
// ~kernel has mirror symmetry along Y and X-axis,
// apart form first row,
// and is only stored (roughly) half:
//
// K11, K22, K02:
// xxxxx
// aaaaa
// bbbbb
// ....
// bbbbb
// aaaaa
//
// K12:
// xxxxx
// aaaaa
// bbbbb
// ...
// -aaaa
// -bbbb

// 3D array indexing

extern "C" __global__ void
kernmulRSymm3D(float* __restrict__  fftMx,  float* __restrict__  fftMy,  float* __restrict__  fftMz,
               float* __restrict__  fftKxx, float* __restrict__  fftKyy, float* __restrict__  fftKzz,
               float* __restrict__  fftKyz, float* __restrict__  fftKxz, float* __restrict__  fftKxy,
               int N0, int N1, int N2) {

    int i = blockIdx.z * blockDim.z + threadIdx.z;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if(i>= N0 || j>= N1 || k>=N2) {
        return;
    }

    float Kxx, Kyy, Kzz, Kxy, Kxz, Kyz;

    int I = N2*(i*N1 + j) + k;
    int e = 2 * I;
    float reMx = fftMx[e  ];
    float imMx = fftMx[e+1];
    float reMy = fftMy[e  ];
    float imMy = fftMy[e+1];
    float reMz = fftMz[e  ];
    float imMz = fftMz[e+1];

    if (j < N1/2 + 1) {
        Kxx = fftKxx[I];
        Kyy = fftKyy[I];
        Kzz = fftKzz[I];
        Kyz = fftKyz[I];
        Kxz = fftKxz[I];
        Kxy = fftKxy[I];
    } else {
        int I2 = N2*(i*N1 + (N1-j)) + k;
        Kxx =  fftKxx[I2];
        Kyy =  fftKyy[I2];
        Kzz =  fftKzz[I2];
        Kyz = -fftKyz[I2]; // !
        Kxz =  fftKxz[I2];
        Kxy = -fftKxy[I2]; // !
    }

    fftMx[e  ] = reMx * Kxx + reMy * Kxy + reMz * Kxz;
    fftMx[e+1] = imMx * Kxx + imMy * Kxy + imMz * Kxz;
    fftMy[e  ] = reMx * Kxy + reMy * Kyy + reMz * Kyz;
    fftMy[e+1] = imMx * Kxy + imMy * Kyy + imMz * Kyz;
    fftMz[e  ] = reMx * Kxz + reMy * Kyz + reMz * Kzz;
    fftMz[e+1] = imMx * Kxz + imMy * Kyz + imMz * Kzz;
}


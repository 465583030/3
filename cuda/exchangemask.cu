#include "hip/hip_runtime.h"
#include "stencil.h"
#include "mask.h"

// Add 1 component of exchange interaction to Beff (Tesla).
// m is normalized.
// mask defines a pre-factor for (Aex / Msat) to allow space-dependent exchange.
// the mask is staggered over half a cell with respect to the magnetization grid,
// and otherwise has the same size.
// mask{X,Y,Z} defines the coupling between neighbors in the {X,Y,Z} direction, respectively.
// maskX[i, j, k] defines the coupling between m[i, j, k-1] and m[i, j k]
// maskY[i, j, k] defines the coupling between m[i, j-1, k] and m[i, j k]
// maskZ[i, j, k] defines the coupling between m[i-1, j, k] and m[i, j k]
// Each time, the zeroth element defines the coupling at the leftmost boundary and is thus unused,
// but would be used in case if periodic boundary conditions.
extern "C" __global__ void
addexchangemask(float* __restrict__ Beff, float* __restrict__ m, 
                float* __restrict__ maskX, float* __restrict__ maskY, float* __restrict__ maskZ,
                float wx, float wy, float wz, int N0, int N1, int N2){

	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int k = blockIdx.y * blockDim.y + threadIdx.y;

	if (j >= N1 || k >= N2){
		return;
	}

	for(int i=0; i<N0; i++){

		int I = idx(i, j, k);
		float B = Beff[I];
		float m0 = m[I];

		float m1 = m[idx(i, j, lclamp(k-1    ))];
		float m2 = m[idx(i, j, hclamp(k+1, N2))];
		float a1 = loadmask(maskX, idx     (i, j, k  ));
		float a2 = loadmask(maskX, idxclamp(i, j, k+1));
		B += wz * (a1*(m1-m0) + a2*(m2-m0));

		m1 = m[idx(i, lclamp(j-1   ), k)];
		m2 = m[idx(i, hclamp(j+1,N1), k)];
		a1 = loadmask(maskY, idx     (i, j,   k));
		a2 = loadmask(maskY, idxclamp(i, j+1, k));
		B += wy * (a1*(m1-m0) + a2*(m2-m0));

		// only take vertical derivative for 3D sim
		if (N0 != 1){
			m1 = m[idx(hclamp(i+1,N0), j, k)];
			m2 = m[idx(lclamp(i-1   ), j, k)];
			a1 = loadmask(maskZ, idx     (i  , j, k));
			a2 = loadmask(maskZ, idxclamp(i+1, j, k));
			B  += wx * (a1*(m1-m0) + a2*(m2-m0));
		}

		Beff[I] = B;
	}
}


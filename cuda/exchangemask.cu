#include "hip/hip_runtime.h"
#include "stencil.h"
#include "mask.h"

// Add 1 component of exchange interaction to Beff (Tesla).
// m is normalized.
// See exchange.go for more details.
extern "C" __global__ void
addexchangemask(float* __restrict__ Beff, float* __restrict__ m,
                float* __restrict__ maskX, float* __restrict__ maskY, float* __restrict__ maskZ,
                float wx, float wy, float wz, int N0, int N1, int N2) {

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (j >= N1 || k >= N2) {
        return;
    }

    for(int i=0; i<N0; i++) {

        int I = idx(i, j, k);
        float B = Beff[I];
        float m0 = m[I];

        float m1 = m[idx(i, j, lclamp(k-1    ))];
        float m2 = m[idx(i, j, hclamp(k+1, N2))];
        float a1 = loadmask(maskZ, idx     (i, j, k  ));
        float a2 = loadmask(maskZ, idxclamp(i, j, k+1));
        B += wz * (a1*(m1-m0) + a2*(m2-m0));

        m1 = m[idx(i, lclamp(j-1   ), k)];
        m2 = m[idx(i, hclamp(j+1,N1), k)];
        a1 = loadmask(maskY, idx     (i, j,   k));
        a2 = loadmask(maskY, idxclamp(i, j+1, k));
        B += wy * (a1*(m1-m0) + a2*(m2-m0));

        // only take vertical derivative for 3D sim
        if (N0 != 1) {
            m1 = m[idx(hclamp(i+1,N0), j, k)];
            m2 = m[idx(lclamp(i-1   ), j, k)];
            a1 = loadmask(maskX, idx     (i  , j, k));
            a2 = loadmask(maskX, idxclamp(i+1, j, k));
            B  += wx * (a1*(m1-m0) + a2*(m2-m0));
        }

        Beff[I] = B;
    }
}

